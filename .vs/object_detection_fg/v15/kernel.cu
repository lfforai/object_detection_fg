#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "npp.h"
#include "nppdefs.h"
#include "nppcore.h"
#include "nppi.h"
#include "npps.h"
#include <iostream>
#include <unordered_map>
#include <string>
#include <windows.h>
#include "base_op.cuh"

using namespace std;

DWORD WINAPI Fun(LPVOID lpParamter)
{
	for (int i = 0; i < 10; i++)
		cout << "A Thread Fun Display!" << endl;
	return 0L;
}

template<class T>
struct x_y_w_dx_dy_dw{
	T* x;//forward input
	T* y;//forward output
	T* w;//w
	T* dx;//backward  output
	T* dy;//backward input
	T* dw;
};

template<class T1,class T2>
class father{
public:
	T1 name;
	T2 value;
    father(T1 a,T2 b):name(a),value(b){
	};
	//~father(){};
};

//template<class T1,class T2>
//unordered_map<string, father<T1,T2>*> un_map;

//�Ѷ����������ļ�¼�ϣ�������op�ļ�¼��
int main()
{   
	//father<string,float>* ok_o=new father<string,float>("luofeng",2.0);
	//unordered_map<string, father<string, float>*> un_map;
	//un_map.insert(std::pair<string,father<string,float>*>{"first", ok_o});
	//father<string, float>* ok_o2 = new father<string, float>("lf", 3.0);
	//un_map.insert(std::pair<string, father<string, float>*>{"second", ok_o2});
	//un_map.erase("second");
	//cout<<"---------------------"<<endl;
	//for (const auto& person : un_map)
	//{
	//	std::cout << person.first << " is " << person.second << std::endl;
	//}
	//
	//HANDLE hThread = CreateThread(NULL, 0, Fun, NULL, 0, NULL);
	//CloseHandle(hThread);
	//for (int i = 0; i < 10; i++)
	//	cout << "Main Thread Display!" << endl;

	constant<int>* op_c = (constant<int> *)malloc(sizeof(constant<int>));
	int *src = (int *)malloc(3 * 6 * 4 * 2 * sizeof(int));
	int dim[4] = { 3,6,4,2 };
	op_c->init(0,4,dim,src);
	graph<int, base_op>* graph_g=new graph<int,base_op>;
	w_dw_now<int>* w_dw_o;
	base_op<int>* baseop=new base_op<int>(op_c,graph_g,"add");
	base_op<int>* baseop1= new base_op<int>(op_c, graph_g,"sub");
	base_op<int>* baseop2 = new base_op<int>(op_c, graph_g,"mul");

	base_op<int>* addop = new base_op<int>(baseop,baseop1,op_c,graph_g,"addop");
	base_op<int>* active = new base_op<int>(baseop, op_c,graph_g,"eule");
	
	op_c->clear();
	for (vector<base_op<int>*>::const_iterator iter = addop->fathers.cbegin(); iter != addop->fathers.cend(); iter++)
	{
		cout << (*iter)->name_of_op << endl;
	}

	//
	//a.insert_v(baseop->name_of_op,baseop);
	//a.insert_v(baseop1->name_of_op, baseop1);
	//a.insert_v(baseop2->name_of_op, baseop2);
	//a.insert_v(addop->name_of_op, addop);
	//a.insert_v(active->name_of_op, active);
	graph_g->allvalue();
    return 0;
}


