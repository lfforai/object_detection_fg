#include "hip/hip_runtime.h"
#pragma once
#include "gpumathtool.cuh"

#define BLOCK_SIZE 128
template <class value_type>
__global__ void add_vector_kernel(int size, value_type *buffA, value_type *buffB)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] =buffA[idx] + buffB[idx];
};

template <>
__global__ void add_vector_kernel<half1>(int size, half1 *buffA, half1 *buffB)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = __hadd(buffA[idx], buffB[idx]);
}

template <class value_type>
void add_vector_gpu(int size, value_type *buffA, value_type *buffB)
{
	int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	add_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA, buffB);
	checkCudaErrors(hipDeviceSynchronize());
};

template void add_vector_gpu<float>(int, float*, float*);
template void add_vector_gpu<double>(int,double*, double*);
template void add_vector_gpu<int>(int, int*, int*);
template void add_vector_gpu<half1>(int, half1*, half1*);

//math function---------------------------------------------------------------
template <class value_type>
__global__ void log_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = logf(buffA[idx]);
};

template <class value_type>  //   aphal^buffA=a^x
__global__ void apowx_vector_kernel(int size, value_type *buffA, value_type aphal)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = powf(aphal,buffA[idx]);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void xpowa_vector_kernel(int size, value_type *buffA, value_type aphal)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = powf(buffA[idx], aphal);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void sqrt_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = sqrtf(buffA[idx]);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void cos_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] =cosf(buffA[idx]);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void sin_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = sinf(buffA[idx]);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void exp_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = expf(buffA[idx]);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void rec_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = fdividef((value_type)1.0,buffA[idx]);
};


template <class value_type>
//A only used by apowx,xpowa
void math_vector_gpu(gpu_math_op math_op,int size, value_type *buffA,value_type  aphal)
{
	gpu_math_op cudnn_op_math;
	int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	switch(math_op){
	case 0:
		cudnn_op_math = CONS_LOG;
		log_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 1:
		cudnn_op_math = CONS_APOWX;
		apowx_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA, aphal);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 2:
		cudnn_op_math = CONS_XPOWA;
		xpowa_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA, aphal);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 3:
		cudnn_op_math = CONS_SQRT;
		sqrt_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 4:
		cudnn_op_math = CONS_COS;
		cos_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 5:
		cudnn_op_math = CONS_SIN;//
		sin_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 6:
		cudnn_op_math = CONS_EXP;//
		exp_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 7:
		cudnn_op_math = CONS_REC;//1/x,x!=0
		rec_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	default:
		cout << "wrong constant<T> math op type " << endl;
	}
};
template void math_vector_gpu<float>(gpu_math_op, int, float*, float);

//dy/dx ,math function-----------------------------------------
template <class value_type>
__global__ void dlog_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = fdividef(value_type(1.0),(buffA[idx]));
};

template <class value_type>  //   aphal^buffA=a^x
__global__ void dapowx_vector_kernel(int size, value_type *buffA, value_type aphal)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = powf(aphal, buffA[idx])*log(aphal);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void dxpowa_vector_kernel(int size, value_type *buffA, value_type aphal)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = powf(buffA[idx], aphal-1)*aphal;
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void dsqrt_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = powf(buffA[idx],-0.5)*0.5;
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void dcos_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = -sinf(buffA[idx]);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void dsin_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = cosf(buffA[idx]);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void dexp_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = expf(buffA[idx]);
};

template <class value_type>  //   buffA^aphal=x^a
__global__ void drec_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = -fdividef(value_type(1.0), buffA[idx] * buffA[idx]);
};

//typedef enum {
//	//hipdnnReduceTensorOp_t
//	CONS_LOG = 0,
//	CONS_APOWX = 1,
//	CONS_XPOWA = 2,
//	CONS_SQRT = 3,
//	CONS_X2 = 4,
//	CONS_COS = 5,
//	CONS_SIN = 6,
//	CONS_EXP = 7,
//} gpu_math_op;

template <class value_type>
//A only used by apowx,xpowa
void dmath_vector_gpu(gpu_math_op math_op, int size, value_type *buffA, value_type  aphal)
{
	gpu_math_op cudnn_op_math;
	int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	switch (math_op) {
	case 0:
		cudnn_op_math = CONS_LOG;
		dlog_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 1:
		cudnn_op_math = CONS_APOWX;
		dapowx_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA, aphal);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 2:
		cudnn_op_math = CONS_XPOWA;
		dxpowa_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA, aphal);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 3:
		cudnn_op_math = CONS_SQRT;
		dsqrt_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 4:
		cudnn_op_math = CONS_COS;
		dcos_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 5:
		cudnn_op_math = CONS_SIN;//
		dsin_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 6:
		cudnn_op_math = CONS_EXP;//only A useed ,eg::5=> -5+1= -4
		dexp_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 7:
		cudnn_op_math = CONS_REC;//1/x,x!=0
		drec_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	default:
		cout << "aaaa wrong constant<T> math op type " << endl;
	}
};

template void dmath_vector_gpu<float>(gpu_math_op, int, float*, float);