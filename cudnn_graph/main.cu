#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "npp.h"
#include "nppdefs.h"
#include "nppcore.h"
#include "nppi.h"
#include "npps.h"
#include <iostream>
#include <unordered_map>
#include <string>
#include <windows.h>
#include "base_op.cuh"
#include <thread>
#include "graph_active_threads.cuh"
#include "test_tool.h"


#include "constant_class.cuh"
#include "weigh_class.cuh"
#include "x_op.cuh"
#include "exp_op.cuh"
#include "sum_op.cuh"
#include "div_op.cuh"
#include "mul_op.cuh"


using namespace std;
int main()
{
	int dim[4] = { 1,1,1,1 };
	float src[1] = { 5.0 };
	float a[1] = { 2.0 };
	float one_v[1] = { 1.0 };
	
	//init graph
	base_op<float>::global_graph = new graph<float, base_op>;
    base_op<float>::global_w_trainable = new graph<float, variable>;
	base_op<float>::global_placehold_constant = new graph<float, constant>;

    //graph_create
	base_op<float>*  two_con= x_op<float>::convert_cons_to_x_op("a", 1, 4, dim, a);
	base_op<float>*  X_var = x_op<float>::convert_var_to_x_op(true,"X", 1,4, dim, src);
	base_op<float>*  one_con = x_op<float>::convert_cons_to_x_op("1", 1, 4, dim, one_v);
    
	//put into ops
	base_op<float>* x_1 = sum_op<float>::getObejct(one_con, 1.0, -1.0, X_var, "1-x|");
	base_op<float>* exp_2 = exp_op<float>::getObejct(X_var,2.0,"2*e^x|");
	base_op<float>* x_1_2_exp = mul_op<float>::getObejct(exp_2,1.0,x_1,"2*(1-x)*e^x|");

	base_op<float>* x_add_1 = sum_op<float>::getObejct(one_con, 1.0, 1.0, X_var, "1+x|");
	base_op<float>* exp_1 = exp_op<float>::getObejct(X_var, 1.0, "1*e^x|");
	base_op<float>* x_add_1_1_exp = mul_op<float>::getObejct(exp_1, 1.0, x_add_1,"(1+x)*e^x");
	base_op<float>* x_2= sum_op<float>::getObejct(two_con, 1.0, 1.0, X_var, "2+x|");

	base_op<float>* x_add_1_1_exp_x_2= sum_op<float>::getObejct(x_add_1_1_exp, 1.0, 1.0, x_2, "2+x+(1+x)*e^x|");

	base_op<float>* last = div_op<float>::getObejct(x_1_2_exp, 1.0, x_add_1_1_exp_x_2,"last");

	//active 
	graph_active<float>* graph_ac=graph_active<float>::getobject(base_op<float>::global_graph);
	graph_ac->forward_start(0);
	//if (two_con->sons.empty());
	//cout << "empty" << endl;
	graph_ac->backward_start(0);
	return 0;
}


