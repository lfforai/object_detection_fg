#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "npp.h"
#include "nppdefs.h"
#include "nppcore.h"
#include "nppi.h"
#include "npps.h"
#include <iostream>
#include <unordered_map>
#include <string>
#include <windows.h>
#include "base_op.cuh"
#include <thread>
#include "graph_active_threads.cuh"
#include "test_tool.h"
#include "sum_op.cuh"

#include "constant_class.cuh"
#include "weigh_class.cuh"
using namespace std;


int main()
{
	//int *src = (int *)malloc(3 * 6 * 4 * 2 * sizeof(int));
	//int dim[4] = { 3,6,4,2 };
	//constant<int>*  con1=constant<int>::getObject("con1",0, 4, dim, src);
	//constant<int>*  con2 = constant<int>::getObject("con2", 0, 4, dim, src);
	//variable<int>*  w1 = variable<int>::getObject(true,"w1", 0, 4, dim, src);
	//variable<int>*  w2 = variable<int>::getObject(true,"w2", 0, 4, dim, src);
	//vector<constant<int>*>* cons_vector = new vector<constant<int>*>;
	//vector<variable<int>*>* w_vector = new vector<variable<int>*>;
	//cons_vector->push_back(con1);
	//cons_vector->push_back(con2);
	//w_vector->push_back(w1);
	//w_vector->push_back(w2);

 //   base_op<int>::global_graph = new graph<int, base_op>;
	//base_op<int>::global_w_trainable = new graph<int, variable>;
	////base_op<int>* addnew =sum_op<int>::getObejct(cons_vector, "addnew");
	////graph<int, base_op>* global_graph = new graph<int, base_op>;
	//base_op<int>* add = base_op<int>::getObejct(cons_vector,"add");
 //   base_op<int>* sub = base_op<int>::getObejct(cons_vector, "sub");
	//base_op<int>* mul = base_op<int>::getObejct(cons_vector, "mul");
	//base_op<int>* log = base_op<int>::getObejct(add,sub,cons_vector,w_vector,"log");
	//base_op<int>* exp = base_op<int>::getObejct(mul, cons_vector, w_vector, "exp");
	//base_op<int>* log_sub = base_op<int>::getObejct(log,sub, cons_vector, w_vector, "log_sub");
	//base_op<int>* log_exp = base_op<int>::getObejct(log, exp, cons_vector, w_vector, "log_exp");
	//base_op<int>* soft_max = base_op<int>::getObejct(log_sub, log_exp, cons_vector, w_vector,  "soft_max");
	//base_op<int>* cross_entry = base_op<int>::getObejct(soft_max, log_exp, cons_vector, "cross_entry");
	//
	////base_op<int>::global_graph->allvalue();
	////active graph
	//graph_active<int>* gc = graph_active<int>::getobject(base_op<int>::global_graph);
	//cout<<"forward start"<<endl;
	//gc->forward_start(1);
	//cout <<"backward start"<< endl;
 //   gc->backward_start(1);

	//varible out
	//base_op<int>::global_w_trainable->allvalue();
	//test();
    
	int dim[4] = {1,1,2,4};
	int dimC[4] = {1,1,3,1};
	float src[8] = { 4.0,1.0,1.0,3.0,1.0,2.0,2.0,3.0 };
	float srcC[4] ={ 3.0,2.0,1.0};
	constant<float>*  A = constant<float>::getObject("A", 1, 4, dim, src);
	constant<float>*  C = constant<float>::getObject("C", 1, 4, dim, src);
	variable<float>*  B = variable<float>::getObject(true,"B", 1, 4, dimC, srcC);
	float bate = 2.0;
	constant<float>* D= A->scala_mul(bate);
	cout<<D->con_name<<endl;
	constant<float>* D1=D->function_tensor(CONS_LOG,1.0,1);
	cout<<D1->con_name<<endl;

	//float ap0=1.0;
	//float ap1=0.0;
	//float beta=0.0;
	//CONSTANT_OP_ADD = 0,
	//	CONSTANT_OP_MUL = 1,
	//	CONSTANT_OP_TENSOR_MIN = 2,
	//	CONSTANT_OP_TENSOR_MAX = 3,
	//	CONSTANT_OP_TENSOR_SQRT = 4,
	//	CONSTANT_OP_TENSOR_NOT = 5,
	//constant<float>::op_math(CONSTANT_OP_ADD,A, B, C,&ap0,&ap0,&beta);

	//hipdnnReduceTensorOp_t
	//  CONS_REDUCE_TENSOR_ADD = 0,
	//	CONS_REDUCE_TENSOR_MUL = 1,
	//	CONS_REDUCE_TENSOR_MIN = 2,
	//	CONS_REDUCE_TENSOR_MAX = 3,
	//	CONS_REDUCE_TENSOR_AMAX = 4,
	//	CONS_REDUCE_TENSOR_AVG = 5,
	//	CONS_REDUCE_TENSOR_NORM1 = 6,
	//	CONS_REDUCE_TENSOR_NORM2 = 7,
	//	CONS_REDUCE_TENSOR_MUL_NO_ZEROS = 8,
	//int* result = (int*)malloc(2*sizeof(int));
	//constant<float>::op_math_reduce(CONS_REDUCE_TENSOR_NORM2, A, C, &ap0, &beta,result);
	//cout<<result[0]<<endl;
	//cout<<result[1]<<endl;
	//constant<float>::add(C, B, C, &ap0, &ap0, &beta);
	//int a;
	return 0;
}


