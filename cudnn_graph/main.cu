#include "hip/hip_runtime.h"
#pragma once
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "npp.h"
#include "nppdefs.h"
#include "nppcore.h"
#include "nppi.h"
#include "npps.h"
#include <iostream>
#include <unordered_map>
#include <string>
#include <windows.h>
#include "base_op.cuh"
#include <thread>
#include "graph_active_threads.cuh"
#include "test_tool.h"
#include  <ctime>

#include "constant_class.cuh"
#include "weigh_class.cuh"
#include "x_op.cuh"
#include "exp_op.cuh"
#include "sum_op.cuh"
#include "div_op.cuh"
#include "mul_op.cuh"
#include "sin_op.cuh"
#include "cos_op.cuh"
#include "reduce_sum_op.cuh"
#include "tf.cuh"


using namespace std;
using namespace cv;

void addr(float& b) {
	float* c = &b;
	cout<<c<<endl;
}

int main()
{
	clock_t startTime, endTime;
	startTime = clock();//��ʱ��ʼ
	//int dimfold[4] = {-1, 1, 1, 1};
	//int dimfold2[4] = { 1, 1, 1, 1 };

	//int dim[4] = { 1,1,1,1 };
	//int dim1[4] = { 1,1,1,3};
	//float src[3] = {5.0,6.0,7.0};
	//float a[1] = { 2.0 };
	//float one_v[1] = { 1.0 };

	//tf<float> tf; 
	//tf.graph_init();

	//base_op<float>* two_con = tf.constantPlaceholder_o("2",1,4,dimfold);
	//base_op<float>* one_con=tf.constant_o("1", 1, 4, dim, one_v);
	//base_op<float>* X    =tf.variable_o(true, "X", 1, 4, dim1, src);
	//base_op<float>* exp = tf.exp(X);

	//base_op<float>* y1=(*((*one_con) - X))*((*two_con)*exp);//2*exp(x)*(1-x)
	//base_op<float>* y2 =*(*X + two_con) + *(*one_con + X)*exp;
	//base_op<float>* last = *(tf.exp(*(*y1 + tf.sin(X)) / (*y2*tf.cos(X))))+tf.sin(X);
	//int dim_c[4] = { 1, 1, 1, 1 };
	//base_op<float>* reduce = tf.cos(tf.reduce_avg(tf.a_power_x(last,2.0),dim_c));
	//
	////set sess
	//graph_active<float>* sess=tf.session();
	////init placeholder
	//vector<constant<float>*>*  v = new vector<constant<float>*>;
	//constant<float>* aa = constant<float>::getObject("2", 1, 4, dim,a);
	//v->push_back(aa);
	//sess->Placeholder_assgin(*v);
	//aa->clear();

	////run sess
	//sess->ward_start(1,0);
	//sess->ward_start(0,1);
	//endTime = clock();
	//
	////output 
	//cout << "The run time is: " << (double)(endTime - startTime) / CLOCKS_PER_SEC << "s" << endl;
	//cout <<"forward::"<<(reduce->y)->x[0] << endl;
	//cout << "forward::" << (last->y)->x[1] << endl;
	//cout << "forward::" << (last->y)->x[2] << endl;
	//cout<<"----------------------------------------- "<<endl;
	//vector<variable<float>*>* list_dw=base_op<float>::global_dw_trainable->getallvalue();
	//for(const auto& e : *list_dw)
	//  {   
	//	for (int i = 0; i < ((variable<float>*) e)->length; i++)
	//	{
	//		cout <<e->var_name<<":" <<((variable<float>*)e)->x[i] << endl;;
	//	}
	//  }
	
	tensor_reduce_test();
	endTime = clock();
	//output 
	cout << "The run time is: " << (double)(endTime - startTime) / CLOCKS_PER_SEC << "s" << endl;
	return 0;
}


