#include "hip/hip_runtime.h"
#pragma once
#include "gpumathtool.cuh"

#define BLOCK_SIZE 128
template <class value_type>
__global__ void add_vector_kernel(int size, value_type *buffA, value_type *buffB)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] =buffA[idx] + buffB[idx];
};

template <>
__global__ void add_vector_kernel<half1>(int size, half1 *buffA, half1 *buffB)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = __hadd(buffA[idx], buffB[idx]);
}

template <class value_type>
void add_vector_gpu(int size, value_type *buffA, value_type *buffB)
{
	int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	add_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA, buffB);
	checkCudaErrors(hipDeviceSynchronize());
};

template void add_vector_gpu<float>(int, float*, float*);
template void add_vector_gpu<double>(int,double*, double*);
template void add_vector_gpu<int>(int, int*, int*);
template void add_vector_gpu<half1>(int, half1*, half1*);
