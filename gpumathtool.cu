#include "hip/hip_runtime.h"
#pragma once
#include "gpumathtool.cuh"

#define BLOCK_SIZE 128
template <class value_type>
__global__ void add_vector_kernel(int size, value_type *buffA, value_type *buffB)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] =buffA[idx] + buffB[idx];
};

template <>
__global__ void add_vector_kernel<half1>(int size, half1 *buffA, half1 *buffB)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = __hadd(buffA[idx], buffB[idx]);
}

template <class value_type>
void add_vector_gpu(int size, value_type *buffA, value_type *buffB)
{
	int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	add_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA, buffB);
	checkCudaErrors(hipDeviceSynchronize());
};

template void add_vector_gpu<float>(int, float*, float*);
template void add_vector_gpu<double>(int,double*, double*);
template void add_vector_gpu<int>(int, int*, int*);
template void add_vector_gpu<half1>(int, half1*, half1*);

//math function
template <class value_type>
__global__ void log_vector_kernel(int size, value_type *buffA)
{
	const int idx = BLOCK_SIZE * blockIdx.x + threadIdx.x;
	if (idx >= size) {
		return;
	}
	buffA[idx] = logf(buffA[idx]);
};


typedef enum {
	//hipdnnReduceTensorOp_t
	CONS_LOG = 0,
	CONS_APOWX = 1,
	CONS_XPOWA = 1,
	CONS_SQRT = 2,
	CONS_X2 = 3,
	CONS_COS = 4,
	CONS_SIN = 5,
	CONS_EXP = 6,
} gpu_math_op;

template <class value_type>
//A only used by apowx,xpowa
void math_vector_gpu(gpu_math_op math_op,int size, value_type *buffA,value_type A)
{
	gpu_math_op cudnn_op_math;
	int grid_size = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	switch (math_op)
	{
	case 0:
		cudnn_op_math = CONS_LOG;
		log_vector_kernel<value_type> << <grid_size, BLOCK_SIZE >> > (size, buffA, buffB);
		checkCudaErrors(hipDeviceSynchronize());
		break;
	case 1:
		cudnn_op_math = CONS_APOWX;
		break;
	case 2:
		cudnn_op_math = CONS_XPOWA;
		break;
	case 3:
		cudnn_op_math = CONS_X2;
		break;
	case 4:
		cudnn_op_math = CONS_COS;//only A useed 2=2^0.5=1.14...
		break;
	case 5:
		cudnn_op_math = CONS_SIN;//only A useed ,eg::5=> -5+1= -4
		break;
	case 6:
		cudnn_op_math = CONS_EXP;//only A useed ,eg::5=> -5+1= -4
		break;
	default:
		cout << "wrong constant<T> math op type " << endl;
	}
};