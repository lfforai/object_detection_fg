#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "npp.h"
#include "nppdefs.h"
#include "nppcore.h"
#include "nppi.h"
#include "npps.h"
#include <iostream>
#include <unordered_map>
#include <string>
#include <windows.h>
#include "base_op.cuh"
#include <thread>
#include "graph_active_threads.cuh"
#include "test_tool.h"
using namespace std;
int main()
{
	constant<int>* op_c = (constant<int> *)malloc(sizeof(constant<int>));
	int *src = (int *)malloc(3 * 6 * 4 * 2 * sizeof(int));
	int dim[4] = { 3,6,4,2 };
	op_c->init(0, 4, dim, src);

	graph<int, base_op>* graph_g = new graph<int, base_op>;
	base_op<int>* add = new base_op<int>(op_c, graph_g, "add");
	base_op<int>* sub = new base_op<int>(op_c, graph_g, "sub");
	base_op<int>* mul = new base_op<int>(op_c, graph_g, "mul");
	base_op<int>* log = new base_op<int>(add, sub, op_c, graph_g, "add_sub");
	base_op<int>* exp = new base_op<int>(mul, op_c, graph_g, "op_mul");
	base_op<int>* simge = new base_op<int>(log, sub,op_c, graph_g, "log_sub");
	base_op<int>* cos = new base_op<int>(exp, log, op_c, graph_g, "log_exp");
	base_op<int>* exten = new base_op<int>(op_c, graph_g, "exten");
	//graph_g->allvalue();

	graph_active<int>* ga = graph_active<int>::getobject(graph_g);
	cout<<"forward start"<<endl;
	ga->forward_start(1);
	cout <<"backward start"<< endl;
	ga->backward_start(1);
	//test();
	return 0;
}

