#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "npp.h"
#include "nppdefs.h"
#include "nppcore.h"
#include "nppi.h"
#include "npps.h"
#include <iostream>
#include <unordered_map>
#include <string>
#include <windows.h>
#include "base_op.cuh"
#include <thread>
#include "graph_active_threads.cuh"
#include "test_tool.h"
#include "sum_op.cuh"

#include "constant_class.cuh"
#include "weigh_class.cuh"
using namespace std;


int main()
{
	int *src = (int *)malloc(3 * 6 * 4 * 2 * sizeof(int));
	int dim[4] = { 3,6,4,2 };
	constant<int>*  con1=constant<int>::getObject("con1",0, 4, dim, src);
	constant<int>*  con2 = constant<int>::getObject("con2", 0, 4, dim, src);
	variable<int>*  w1 = variable<int>::getObject("w1", 0, 4, dim, src);
	variable<int>*  w2 = variable<int>::getObject("w2", 0, 4, dim, src);
	vector<constant<int>*>* cons_vector = new vector<constant<int>*>;
	vector<variable<int>*>* w_vector = new vector<variable<int>*>;
	cons_vector->push_back(con1);
	cons_vector->push_back(con2);
	w_vector->push_back(w1);
	w_vector->push_back(w2);

	//base_op<int>::init_global_graph_ac_varible<int>();
    base_op<int>::global_graph = new graph<int, base_op>;
	base_op<int>::global_w_trainable = new graph<int, variable>;
	base_op<int>* addnew =sum_op<int>::getObejct(cons_vector, "addnew");
	//graph<int, base_op>* global_graph = new graph<int, base_op>;
	base_op<int>* add = base_op<int>::getObejct(cons_vector,"add");
    base_op<int>* sub = base_op<int>::getObejct(cons_vector, "sub");
	base_op<int>* mul = base_op<int>::getObejct(cons_vector, "mul");
	base_op<int>* log = base_op<int>::getObejct(add,sub,cons_vector,w_vector,"log");
	base_op<int>* exp = base_op<int>::getObejct(mul, cons_vector, w_vector, "exp");
	base_op<int>* log_sub = base_op<int>::getObejct(log,sub, cons_vector, w_vector, "log_sub");
	base_op<int>* log_exp = base_op<int>::getObejct(log, exp, cons_vector, w_vector, "log_exp");
	base_op<int>* soft_max = base_op<int>::getObejct(log_sub, log_exp, cons_vector, w_vector,  "soft_max");
	base_op<int>* cross_entry = base_op<int>::getObejct(soft_max, log_exp, cons_vector, "cross_entry");
	
	//base_op<int>::global_graph->allvalue();
	//active graph
	graph_active<int>* gc = graph_active<int>::getobject(base_op<int>::global_graph);
	cout<<"forward start"<<endl;
	gc->forward_start(1);
	cout <<"backward start"<< endl;
    gc->backward_start(1);

	//varible out
	base_op<int>::global_w_trainable->allvalue();
	//test();
	addnew->backward_function();
	return 0;
}


