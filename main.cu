#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "npp.h"
#include "nppdefs.h"
#include "nppcore.h"
#include "nppi.h"
#include "npps.h"
#include <iostream>
#include <unordered_map>
#include <string>
#include <windows.h>
#include "base_op.cuh"
#include <thread>
#include "graph_active_threads.cuh"
#include "test_tool.h"
#include  <ctime>

#include "constant_class.cuh"
#include "weigh_class.cuh"
#include "x_op.cuh"
#include "exp_op.cuh"
#include "sum_op.cuh"
#include "div_op.cuh"
#include "mul_op.cuh"
#include "sin_op.cuh"
#include "cos_op.cuh"
#include "tf.cuh"

using namespace std;

void addr(float& b) {
	float* c = &b;
	cout<<c<<endl;
}

int main()
{
	clock_t startTime, endTime;
	startTime = clock();//��ʱ��ʼ
	int dim[4] = { 1,1,1,1 };
	int dim1[4] = { 1,1,1,3};
	float src[3] = {5.0,6.0,7.0};
	float a[1] = { 2.0 };
	float one_v[1] = { 1.0 };

	tf<float> tf; 
	tf.graph_init();
	
	/*base_op<float>* p = tf.constantPlaceholder_o("px",1,4,dim);*/
	base_op<float>* two_con=tf.constant_o("2", 1, 4, dim, a);
	base_op<float>* one_con=tf.constant_o("1", 1, 4, dim, one_v);
	base_op<float>* X    =tf.variable_o(true, "X", 1, 4, dim1, src);
	base_op<float>* exp = tf.exp(X);

	base_op<float>* y1=(*((*one_con) - X))*((*two_con)*exp);//2*exp(x)*(1-x)
	base_op<float>* y2 =*(*X + two_con) + *(*one_con + X)*exp;
	base_op<float>* last = *(tf.exp(*(*y1 + tf.sin(X)) / (*y2*tf.cos(X))))+tf.sin(X);
	
	graph_active<float>* graph_ac=graph_active<float>::getobject(base_op<float>::global_graph);
	graph_ac->ward_start(0,0);
	cout << "---------------------------------------" << endl;
	graph_ac->ward_start(0,1);
	endTime = clock();
	cout <<"--------------------------------------- "<<endl;
	cout << "The run time is: " << (double)(endTime - startTime) / CLOCKS_PER_SEC << "s" << endl;
	cout <<"forward::"<<(last->y)->x[0] << endl;
	cout << "forward::" << (last->y)->x[1] << endl;
	cout << "forward::" << (last->y)->x[2] << endl;
	cout<<"----------------------------------------- "<<endl;
	cout<<"backward::"<< ((variable<float>*)(X->dw))->x[0]<<endl;
	cout << "backward::" << ((variable<float>*)(X->dw))->x[1] << endl;
	cout << "backward::" << ((variable<float>*)(X->dw))->x[2] << endl;
	return 0;
}


